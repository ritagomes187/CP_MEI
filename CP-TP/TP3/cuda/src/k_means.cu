#include "hip/hip_runtime.h"
// 1 Passo : incializar os pontos e centroids dos clusters
// 2 Passo: associar pontos aos clusters pro centroid mais perto,  calculando distancia euclidiana entre o ponto e o centroid do cluster
// 3 Passo: recalcular centroide fazendo média dos pontos associados
// 4 Passo: 2 e 3 até que não haja mais trocas de clusters

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "/usr/include/linux/hip/hip_runtime.h"

#include "../include/k_means.h"

// #define N 0

// #define K 0


// Inicialização dos clusters e pontos
void inicializa(Points *p, Points *clust, int N, int K)
{
    float x,y;
    srand(10);
    // inicializa os pontos
    for(int i = 0; i < N;) {
        x = (float) rand() / RAND_MAX;
        y = (float) rand() / RAND_MAX;
        p[i].x = x;
        p[i].y = y; 
        
        x = (float) rand() / RAND_MAX;
        y = (float) rand() / RAND_MAX;
        p[i+1].x = x;
        p[i+1].y = y;
        i+=2;

    }
    
    //incializa clusters
    for(int j = 0; j < K;) {
        clust[j] = p[j];
        clust[j+1] = p[j+1];
        j+=2;
        
    }
}
// Calcula distância euclidiana entre dois pontos
//  float calculaDist(Points p1, Points p2)
// {
//     float y = p1.x - p2.x;
//     float x = p1.y - p2.y;

//     float dist = sqrt(pow(y, 2) + pow(x, 2));

//     return dist;
// }

// Inicializar a associação dos pontos aos clusters mais próximos
__global__ void associaPontosInit(Points *cluster, Points *pontos, int N, int K)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N)
    {
        int clusterMin, y;
        float distMin;
        float *distancias = (float*) malloc(K * sizeof(float)); // vetor que irá conter as distancias euclidianas entre cada ponto e os centroids
        Points ponto;
        distMin = 10000;
        clusterMin = 0;
        ponto = pontos[i];
        // local de paralelismo
        for (y = 0; y < K; y++)
        { 
            float z = ponto.x - cluster[y].x;
            float x = ponto.y - cluster[y].y;

            float dist = sqrt(pow(z, 2) + pow(x, 2));
            distancias[y] = dist;
            
        }
        for (y = 0; y < K; y++)
        {
            if (distancias[y] < distMin)
            {
                distMin = distancias[y];
                clusterMin = y;
            }
        }
        pontos[i].idC_size = clusterMin;
        atomicAdd(&cluster[clusterMin].idC_size, 1);
        
    }
    
}
// Associar os pontos aos clusters mais próximos
__global__ void associaPontos(Points *cluster, Points *pontos, int N, int K)
{
    // int flag = 0;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N)
    {
        int clusterAntigo, clusterIdMin, j;
        float distMin;
        Points ponto, centroidMin;
        
        float *distancias = (float*) malloc(K * sizeof(float)); // vetor que irá conter as distancias euclidianas entre cada ponto e os centroids 
        ponto = pontos[i];
        float z,x,dist;
        for (j = 0; j < K; j++)
        {
             z = ponto.x - cluster[j].x;
             x = ponto.y - cluster[j].y;

            dist = sqrt(pow(z, 2) + pow(x, 2));
            distancias[j] = dist;
        }
        clusterIdMin = ponto.idC_size;
        clusterAntigo = clusterIdMin;
        centroidMin = cluster[clusterAntigo];
        
        z = ponto.x - centroidMin.x;
        x = ponto.y - centroidMin.y;
        dist = sqrt(pow(z, 2) + pow(x, 2));
        distMin = dist;
        for (j = 0; j < K; j++)
        {
            if (distancias[j] < distMin)
            {
                distMin = distancias[j];
                clusterIdMin = j;
            }
        }
        // realizar troca de cluster
        if (clusterIdMin != clusterAntigo)
        {
            // flag=1;
            pontos[i].idC_size = clusterIdMin;

            atomicAdd(&cluster[clusterAntigo].idC_size, -1);
            atomicAdd(&cluster[clusterIdMin].idC_size, 1);
        }
        free(distancias);
    }

    // return flag;
}

void inicializaVectors(float acumulaX[],float acumulaY[],int K){
     // inicializa vetores que irão ter a soma dos pontos
    for (int z = 0; z<K;z++){
        acumulaX[z] = 0.0;
        acumulaY[z] = 0.0;
    }
}

    

// Recalcula centroid de cada cluster
__global__ void associaCentroid(Points *cluster ,int K,float acumulaX[], float acumulaY[]){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Faz a media dos pontos e associa a um novo centroid a cada cluster
    if (i < K)
    {
        int size;
        float mediaX, mediaY;
        size = cluster[i].idC_size;
        mediaX = acumulaX[i] / (float)size;
        mediaY = acumulaY[i] / (float)size;
        // ponto de paralelismo
        cluster[i].x = mediaX;
        cluster[i].y = mediaY;
    }

}


__global__ void recalculaCentroid(Points *cluster, Points *pontos, int N, float acumulaX[], float acumulaY[])
{
    int y = threadIdx.x + blockIdx.x * blockDim.x;
/* realiza soma dos pontos */
    if (y < N)
    {
        int clustId;
        Points ponto;
        ponto = pontos[y];
        clustId = ponto.idC_size;
        atomicAdd(&acumulaX[clustId], ponto.x);
        atomicAdd(&acumulaY[clustId], ponto.y);
    }
    
    
}

int main(int argc, char *argv[])
{
    int N, K;
    N = atoi(argv[1]);
    K = atoi(argv[2]);
    int threadsPerBlock = 256;
    int blocksPerGrid =  (N + threadsPerBlock - 1) / threadsPerBlock;
    
    float acumulaX[K];
    float acumulaY[K];
    Points *d_pontos;
    Points *d_cluster;
    // O hipMalloc pode ser usado para alocar memória na GPU para os dados que serão processados pelo seu kerne
    //  Aloca memória para o array pontos
    hipMalloc(&d_pontos, N * sizeof(Points));
    
    hipMalloc(&d_cluster, K * sizeof(Points));
    
    // Aloca memória para o array cluster
    

    float *d_acumulaX, *d_acumulaY;
    hipMalloc(&d_acumulaX, K * sizeof(float));
    hipMalloc(&d_acumulaY, K * sizeof(float));


    Points *pontos = NULL;
    Points *cluster = NULL;
    pontos = (Points*) malloc(N * sizeof(Points));
    cluster = (Points*) malloc(K * sizeof(Points));
    inicializa(pontos, cluster, N, K);
    

    // Copia os dados dos arrays na CPU para os arrays na GPU


    // Executa o kernel
    // o segundo argumento <<<1,1>>> especifica o número de threads em cada dimensão do grid
    // e o número de threads em cada dimensão do bloco

    // Copia os dados dos arrays na GPU de volta para os arrays na CPU
    // int flag = 1;
    int it = 0;
    //inicializa(p<<<blocksPerGrid,threadsPerBlock>>>ontos, clust, N, K);
    
    
    hipMemcpy(d_pontos, pontos, N * sizeof(Points), hipMemcpyHostToDevice);
    hipMemcpy(d_cluster, cluster, K * sizeof(Points), hipMemcpyHostToDevice);
    
    hipError_t err;
    associaPontosInit<<<blocksPerGrid,threadsPerBlock>>>(d_cluster, d_pontos, N, K);
    err = hipMemcpy(cluster, d_cluster, K * sizeof(Points), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error3: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    Points ponto;
    for (int j = 0; j < K; j++)
    {
        ponto = cluster[j];
        printf("Center: (%f,%f) : Size: %d\n", ponto.x, ponto.y, ponto.idC_size);
    }

    //hipMemcpy(cluster, d_cluster, K * sizeof(Points), hipMemcpyDeviceToHost);
    while (it < 20)
    {
        
        inicializaVectors(acumulaX, acumulaY, K);
        hipMemcpy(d_acumulaX, acumulaX, K * sizeof(Points), hipMemcpyHostToDevice);
        hipMemcpy(d_acumulaY, acumulaY, K * sizeof(Points), hipMemcpyHostToDevice);
    
        recalculaCentroid<<<blocksPerGrid,threadsPerBlock>>>(d_cluster, d_pontos, N,d_acumulaX, d_acumulaY);
        associaCentroid<<<blocksPerGrid,threadsPerBlock>>>(d_cluster ,K,d_acumulaX, d_acumulaY);
        associaPontos<<<blocksPerGrid,threadsPerBlock>>>(d_cluster, d_pontos, N, K);
        
        //printf("oi\n");
        //ponto = cluster[0];
        //printf("Center: (%f,%f) : Size: %d\n", ponto.x, ponto.y, ponto.idC_size);
        it++;
    }
    printf("N = %d, K = %d\n", N, K);
    err = hipMemcpy(cluster, d_cluster, K * sizeof(Points), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    for (int j = 0; j < K; j++)
    {
        ponto = cluster[j];
        printf("Center: (%f,%f) : Size: %d\n", ponto.x, ponto.y, ponto.idC_size);
    }
    printf("Iterations: %d\n", it);
    freePoints(pontos);
    freePoints(cluster);

    // Liberta a memória da GPU quando não for mais necessária
    hipFree(d_pontos);
    hipFree(d_cluster);
    hipFree(d_acumulaX);
    hipFree(d_acumulaY);
    return 0;
}